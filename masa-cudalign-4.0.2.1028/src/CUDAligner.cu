#include "hip/hip_runtime.h"
/*******************************************************************************
 *
 * Copyright (c) 2010-2015   Edans Sandes
 *
 * This file is part of MASA-CUDAlign.
 * 
 * MASA-CUDAlign is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * MASA-CUDAlign is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with MASA-CUDAlign.  If not, see <http://www.gnu.org/licenses/>.
 *
 ******************************************************************************/

#include "CUDAligner.hpp"
#include "cuda_util.h"
#include "config.h"

#include <stdio.h>
#include <getopt.h>

/**
 * @file CUDAligner.cu
 * @brief CUDAligner related kernels and wrappers.
 *
 * This file contains all the kernels and some wrappers for the
 * CUDAligner class. Ensure that all CUDAligner.* files are compiled
 * with the same number of threads.
 *
 * The function templates accelerate the computation using precompiled
 * different kernels. We have four templates: COLUMN_SOURCE can be 0 (first
 * column is zeroed) or 1 (first column is customized vector);
 * COLUMN_DESTINATION can be 0 (last
 * column is discarded) or 1 (last column is stored in a vector);
 * CHECK_LOCATION can be 0 (ignore best scores) or 1 (check for best scores);
 * and RECURRENCE_TYPE can be SMITH_WATERMAN or NEEDLEMAN_WUNSCH.
 *
 * @remark This file must be compiled by nvcc NVidia compiler.
 */


/** First column must be zeroes */
#define FROM_ZEROES			(0)
/** First column must be loaded with custom data from a vector. */
#define FROM_VECTOR			(1)


/** Discard last column cells */
#define DISCARD_LAST_COLUMN			(false)
/** Stores tha last column cells in a vector */
#define STORE_LAST_COLUMN			(true)


/** Check best score in the block cells. */
#define CHECK_BEST_SCORE			(true)
/** Do not check best score in the block cells. */
#define IGNORE_BEST_SCORE			(false)


/*
 * Texture definition for storing read-only large variables.
 */

/**
 * Texture definition for the DNA of sequence#0 (vertical in the DP matrix).
 */
texture<unsigned char, 1, hipReadModeElementType> t_seq0;

/**
 * Texture definition for the DNA of sequence#1 (horizontal in the DP matrix).
 */
texture<unsigned char, 1, hipReadModeElementType> t_seq1;


/**
 * Texture definition for the read-only part of the horizontal bus.
 * This is a tricky part, since the same horizontal bus is used for
 * writing and for reading. The data is read from the horizontal bus
 * in a linear pattern (left to right) using the texture. The data
 * is written in the same horizontal bus using the same linear pattern
 * (left to right), but the written data is shifted left some cells
 * compared with the reading data. So, the written data does not affect the
 * read-only data. In every external diagonal we clean the texture cache
 * for safety. For the single phase execution (small sequences), this
 * optimization must be disabled.
 */
texture<         int2, 1, hipReadModeElementType> t_busH;



/*
 * Shared Memory structures, used for sharing data between the threads of
 * the same block.
 */

/**
 * Used to share the H data of the Smith-Waterman cell. We have a
 * double buffer structure to allow simultaneous non-block read-write.
 */
__shared__   int  s_colx [2][THREADS_COUNT]; // Could be THREADS_COUNT-1

/**
 * Used to share the F data of the Smith-Waterman cell. We have a
 * double buffer structure to allow simultaneous non-block read-write.
 */
__shared__   int  s_coly [2][THREADS_COUNT];

/**
 * Stores the column position that splits each block, e.g., block #0 processes
 * columns between d_split[0](inclusive) and d_split[1](exclusive). Generally,
 * block $i$ processes columns between d_split[i-1] and d_split[i].
 */
__constant__ int d_split[MAX_BLOCKS_COUNT+1];



/**
 * Returns true only to the thread with the maximum value. This function is
 * much faster than the findMaxSmall function, but it only works if the number
 * of threads is known in compilation time and if the number of threads is
 * a power of 2. So, do not use this function for generic number of threads.
 *
 * @param idx the thread Id.
 * @param val the value that will be compared
 * @return true to the thread with the maximum value. False otherwise. If
 *              more than one thread has the maximum value, only one of them
 *              will return true.
 */
__device__ int findMax(int idx, int val) {
    __shared__ int s_max[THREADS_COUNT];
    __shared__ int s_idx;
    int count = THREADS_COUNT>>1;
    s_max[idx] = val;
    __syncthreads();

    while (count > 0) {
        if (idx < count) {
            if (s_max[idx] < s_max[idx+count]) {
                s_max[idx] = s_max[idx+count];
            }
        }
        count = count >> 1;
        __syncthreads();
    }
    if (s_max[0] == val) {
        s_idx = idx;
    }
    __syncthreads();
    return (s_idx == idx);
}

/**
 * Returns true only to the thread with the maximum value. This function is
 * slower then the findMax function, but it works for a variable number of
 * threads. This should be used in the kernel with a single block, since
 * the number of threads cannot be greater than the size of sequence #2
 * (horizontal sequence).
 *
 * @param idx the thread Id.
 * @param val the value that will be compared
 * @return true to the thread with the maximum value. False otherwise. If
 *              more than one thread has the maximum value, only one of them
 *              will return true.
 */
__device__ int findMaxSmall(int idx, int val) {
	__shared__ int s_max;
	__shared__ int s_idx;

	s_max = -INF;
	__syncthreads();

	atomicMax(&s_max, val);
	if (s_max == val) {
		s_idx = idx;
	}
    __syncthreads();
    return (s_idx == idx);
}

/**
 * Consider that each thread processes ALPHA rows, so the last row of the
 * matrix may be any of the ALPHA rows of the last thread. Sometimes, there
 * will be some rows of the thread that will not be processed. Furthermore,
 * the last row of the DP matrix must be flushed to the disk in many situations,
 * so, the very last thread of a block must save all its data in different
 * structures.
 *
 * This function returns information about which rows of the thread must
 * be processed, considering that the DP matrix must be fully processed
 * between rows $i0$ and $i1$.
 *
 * @param i the number of the first row (top) of the thread.
 * @param i0 the number of the first row (top) of the processed area
 *           of the DP matrix (usually 0 in stage 1).
 * @param i1 the number of the last row (bottom) of the processed area
 *           of the DP matrix (usually the length of seq0 in stage 1).
 *
 * @return Possible return values are:
 *   0: if all the rows of the thread is out of range [i0..i1].
 *   -1: if all the rows of the thread is in the range [i0..i1].
 *   1: if only the first row [1] of the thread is in the range [i0..i1].
 *   2: if only the rows [1,2] of the thread are in the range [i0..i1].
 *   3: if only the rows [1,2,3] of the thread are in the range [i0..i1].
 */
static __device__ int get_flush_id(const int i, const int i0, const int i1) {
    if (i < i0 || i >= i1) {
    	return 0;
    } else {
    	return (i+ALPHA < i1) ? -1 : ((i1-i0-1)%ALPHA)+1;
    }
}

/**
 * Returns the maximum of two numbers.
 */
static __device__ int my_max(int a, int b) {
    return (a>b)?a:b;
}

/**
 * Returns the maximum of three numbers.
 */
static __device__ int my_max3(int a, int b, int c) {
    return my_max(a, my_max(b,c));
}

/**
 * Returns the maximum of four numbers.
 */
static __device__ int my_max4(int a, int b, int c, int d) {
    return my_max(my_max(a,b), my_max(c,d));
}

/**
 * Fetch a block with 4 nucleotides at positions [i..i+3] of sequence t_seq0.
 *
 * @param i the row of the first base of the DNA 4-block.
 *        The i index is 0-based.
 * @return ss the 4 bases in the (x,y,z,w) uchar4 vector, representing
 *         the bases (i, i+1, i+2, i+3) of the sequence t_seq0.
 */
static __device__ void fetchSeq0(const int i, uchar4* ss) {
	if (i >= 0) {
        ss->x = tex1Dfetch(t_seq0,i);
        ss->y = tex1Dfetch(t_seq0,i+1);
        ss->z = tex1Dfetch(t_seq0,i+2);
        ss->w = tex1Dfetch(t_seq0,i+3);
	}
}


/**
 * This procedure computes the cell (i,j) using the Smith-Waterman recurrence
 * formula or the Needleman-Wunsch recurrence formula. Consider that the
 * we use the affine-gap model, so the Dynamic Programming (DP) matrix is
 * compose of cells with 3 components (H, E, F). See the Gotoh algorithm
 * for better comprehension.
 *
 * @tparam RECURRENCE_TYPE	Template for SMITH_WATERMAN or NEEDLEMAN_WUNSCH.
 * @param[in] s0	The variable containing seq0[i] nucleotide.
 * @param[in] s1	The variable containing seq1[i] nucleotide.
 * @param[in,out] e00	Input: value of E(i,j-1). Output: value of E(i,j)
 * @param[in,out] f00	Input: value of F(i,j-1). Output: value of F(i,j)
 * @param[in] h01	value of H(i,j-1)
 * @param[in] h11	value of H(i-1,j-1)
 * @param[in] h10	value of H(i-1,j)
 * @param[out] h00  value of H(i,j)
 */
template <int RECURRENCE_TYPE>
__device__ void kernel_sw(const unsigned char s0, const unsigned char s1,
						  int *e00, int *f00, const int h01, const int h11, const int h10, int *h00) {

    *e00 = my_max(h01-DNA_GAP_OPEN, *e00)-DNA_GAP_EXT; // Horizontal propagation
    *f00 = my_max(h10-DNA_GAP_OPEN, *f00)-DNA_GAP_EXT; // Vertical propagation
    int v1 = h11+((s1!=s0)?DNA_MISMATCH:DNA_MATCH);

    if (RECURRENCE_TYPE == SMITH_WATERMAN) {
    	*h00 = my_max4(0, v1, *e00, *f00);
    } else if (RECURRENCE_TYPE == NEEDLEMAN_WUNSCH) {
    	*h00 = my_max3(v1, *e00, *f00);
    }
}

/**
 * Updates the maximum score found ($max$) and its position ($pos$) after
 * the computation of the cell at position (i+inc,j). The $i+inc$ coordinate
 * considers that each thread computes $ALPHA$ cells, so we split the
 * coordinate in $i$ and $inc$, where $i$ is the first coordinate of the ALPHA
 * cells and $inc$ increases the coordinate of the following cells. When the
 * $absolute_row$ parameter is true, the best score position ($pos$) is
 * updated with the absolute coordinate $i+inc$, but if $absolute_row$ is false,
 * then $pos$ is updated with the incremental $inc$ value, avoiding one add
 * operation. In this case, the missing add operation is done only once
 * in the end of the kernel execution, leading to better performance.
 *
 * @param[in] i			The index of the row.
 * @param[in] j			The index of the column.
 * @param[in,out] max 	Input: previous maximum score.
 * 						Output: updated maximum score.
 * @param[in,out] pos 	Input: coordinates of the previous maximum score.
 * 								See the $absolute_row$ parameter.
 *            			Output: updated coordinates of the maximum score.
 * @param[in] h00 	The score of cell (i+inc,j). Only the H component is necessary,
 *                   	since the other components E and F cannot be greater than H.
 * @param[in] inc 	This is the row id inside each thread neighborhood.
 * @param[in] absolute_row 	If it is true, the $pos$ parameter stores the absolute
 *                   coordinates of the cell (i+inc, j). Otherwise, if
 *                   $store$ is false, the $pos$ parameters stores the relative
 *                   coordinates of the cell (inc, j).
 */
__device__ void kernel_check_max(const int i, const int j, int *max,
		int2 *pos, const int h00, const int inc, const bool absolute_row) {
	if (*max < h00) {
		*max = h00;
		pos->x = j;
		if (absolute_row) {
			pos->y = i+inc;
		} else {
			pos->y = inc;
		}
	}
}

/**
 * Computes a block of ALPHA cells inside the same thread (K-neighborhood).
 * The computation is done serially (up-down). The processed cells are
 * (i,j), (i+1,j), (i+2,j), (i+3,j), considering that ALPHA is 4.
 *
 * @tparam RECURRENCE_TYPE	Defines with recurrence function will be use. It
 * 								can be SMITH_WATERMAN (local alignment) or
 * 								NEEDLEMAN_WUNSCH (global alignment).
 * @tparam FLUSH_LAST_ROW	Defines if the flush_id parameter will be used.
 * 								If it is false,	all the ALPHA rows will be
 * 								processed regardless if they are out-of-range.
 * 								Otherwise, only valid rows will be processed.
 *
 * @param[in] i		The index of the row (top cell).
 * @param[in] j		The index of the column.
 * @param[in] s1	One nucleotide related to sequence#1 (horizontal).
 * @param[in] ss	A block with 4 (ALPHA) nucleotides related to
 * 						sequence#0 (vertical).
 * @param[in,out] left_e	Input: Values of E(i,j-1)...E(i+3,j-1)
 * 							Output: Values of E(i,j)...E(i+3,j)
 * @param[in,out] up_f		Input: Value of F(i-1,j)
 * 							Output: Value of F(i+3,j)
 * @param[in] left_h		Input: Value of H(i,j-1)...H(i+3,j-1)
 * @param[in] diag_h		Input: Value of H(i-1,j-1)
 * @param[in] up_h			Input: Value of H(i-1,j)
 * @param[out] curr_h		Output: Value of H(i,j)...H(i+3,j)
 * @param[in] flush_id		Identifier of the last row. This value controls with
 * 								row will be computed in the bottom-most blocks.
 * 								See get_flush_id function.
 *
 */
template <int RECURRENCE_TYPE, bool FLUSH_LAST_ROW>
__device__ void kernel_sw4(const int i, const int j, const unsigned char s1, const uchar4 ss,
									int4 *left_e, int *up_f, const int4 left_h, const int diag_h, const int up_h, int4 *curr_h, const int flush_id) {
	kernel_sw<RECURRENCE_TYPE>(s1, ss.x, &left_e->x, up_f, left_h.x, diag_h  , up_h  , &curr_h->x);
	if (FLUSH_LAST_ROW && flush_id == 1) return;
	kernel_sw<RECURRENCE_TYPE>(s1, ss.y, &left_e->y, up_f, left_h.y, left_h.x, curr_h->x, &curr_h->y);
	if (FLUSH_LAST_ROW && flush_id == 2) return;
	kernel_sw<RECURRENCE_TYPE>(s1, ss.z, &left_e->z, up_f, left_h.z, left_h.y, curr_h->y, &curr_h->z);
	if (FLUSH_LAST_ROW && flush_id == 3) return;
	kernel_sw<RECURRENCE_TYPE>(s1, ss.w, &left_e->w, up_f, left_h.w, left_h.z, curr_h->z, &curr_h->w);
}

/**
 * Updates the maximum score of a block of ALPHA cells (K-neighborhood).
 *
 * @tparam CHECK_MAX_SCORE	Defines if the function will be executed. If it is false,
 * 								this function does absolutely nothing.
 * @tparam FLUSH_LAST_ROW	Defines if the flush_id parameter will be used.
 * 								If it is false,	all the ALPHA rows will be
 * 								processed regardless if they are out-of-range.
 * 								Otherwise, only valid rows will be processed.
 *
 * @param[in] i		The index of the row (top cell).
 * @param[in] j		The index of the column.
 * @param[in,out] max		The maximum score to be updated.
 * @param[in,out] max_pos	The coordinates of the maximum score.
 * @param[in] curr_h		The values of H(i,j)...H(i+3,j)
 * @param[in] absolute_row 	Defines if the row coordinates (i) stored in the max_pos
 * 							parameter must be absolute or relative.
 * 							If it is relative (false), the coordinates row may only be in
 * 							the range 0..3. If it is absolute (true), the coordinates
 * 							row is $i+inc$.
 * @param[in] flush_id		Identifier of the last row. See get_flush_id function.
 */
template <bool CHECK_MAX_SCORE, bool FLUSH_LAST_ROW>
__device__ void kernel_check_max4(const int i, const int j,
						  int *max, int2 *max_pos,
						  int4 *curr_h,
						  const bool absolute_row, const int flush_id) {
	if (CHECK_MAX_SCORE) {
		kernel_check_max(i, j, max, max_pos, curr_h->x, 0, absolute_row);
		if (FLUSH_LAST_ROW && flush_id == 1) return;
		kernel_check_max(i, j, max, max_pos, curr_h->y, 1, absolute_row);
		if (FLUSH_LAST_ROW && flush_id == 2) return;
		kernel_check_max(i, j, max, max_pos, curr_h->z, 2, absolute_row);
		if (FLUSH_LAST_ROW && flush_id == 3) return;
		kernel_check_max(i, j, max, max_pos, curr_h->w, 3, absolute_row);
	}
}

/**
 * This function loads the dependencies H(i-1,j) and F(i-1,j) before computing
 * cells (i,j)..(i,j+3). Furthermore, the nucleotide of column $j$ is read
 * from sequence#1 (horizontal).
 *
 * The values H(i-1,j) and F(i-1,j) may be read from
 * the shared memory or from the horizontal bus, depending if $i$ is the
 * first row of the block or not. If USE_TEXTURE_CACHE is used, the
 * horizontal bus is read from a texture cache (t_busH). See the t_busH comments
 * for a better comprehension of this optimization and its pitfall.
 * For very small sequences, this optimization does not work and the
 * horizontal bus is read directly from the busH global vector.
 *
 * @tparam USE_TEXTURE_CACHE	Enables or disable the t_busH texture read. If
 * 									it is disabled, the read is made directly
 * 									from the original busH global vector.
 *
 * @param[in] idx	Index of the current thread.
 * @param[in] bank		The shared memory bank (1 or 0) from which the data will be taken.
 *						This is necessary for the double buffer used in the shared memory.
 *						If bank is 0, the data is read from bank 0 and written in bank 1.
 *						If bank is 1, the data is read from bank 1 and written in bank 0.
 * @param[in] j		The index of the column.
 * @param[in] busH		Horizontal bus. This is the global vector from which we
 * 						load the H(i-1,j) and F(i-1,j) values for the first thread.
 * @param[out] h	The loaded value of H(i-1,j)
 * @param[out] f	The loaded value of F(i-1,j)
 * @param[out] s	The j-th dna nucleotide of sequence#1 (horizontal)
 */
template <bool USE_TEXTURE_CACHE>
__device__ void kernel_load(const int idx, const int bank, const int j, int2* busH, int *h, int *f, unsigned char *s) {
	*s = tex1Dfetch(t_seq1, j);
    if (idx) {
    	// Threads (except the first one) must read from the shared memory.
        *h = s_colx[bank][idx];
        *f = s_coly[bank][idx];
    } else {
    	// First thread of the block must read from the horizontal bus.
        int2 temp;
        if (USE_TEXTURE_CACHE) {
        	temp = tex1Dfetch(t_busH,j); // read from texture
        } else {
        	temp = busH[j]; // read directly from the busH global vector
        }
        *h = temp.x; // H-component
        *f = temp.y; // F-component
    }
}


/**
 * This function writes the values H(i+3,j) and F(i+3,j) of the thread into the
 * horizontal bus or into the shared memory. With this procedure, the next
 * thread can load its dependencies in the next internal diagonal. See function
 * kernel_load.
 *
 * If the thread has the last row of the matrix, this row may be saved in
 * the busH/extraH vectors for further usage. Otherwise, the busH structure
 * may contain values of an out-of-range row (since it may not be multiple of 4
 * and it may be unaligned with the last row of the block).
 *
 * @tparam  FLUSH_LAST_ROW		Indicates if the last row of the matrix must be
 * 									stored in the busH/extraH vectors
 *
 * @param[in] i		The index of the row (top cell).
 * @param[in] j		The index of the column.
 * @param[in] idx	Index of the current thread.
 * @param[in] bank	The shared memory bank (1 or 0) into which the data will be stored.
 *						This is necessary for the double buffer used in the shared memory.
 *						If bank is 1, the data is read from bank 0 and written in bank 1.
 *						If bank is 0, the data is read from bank 1 and written in bank 0.
 * @param[out] busH		Horizontal bus. This is the global vector that receives the
 * 						H(i+3,j) and F(i+3,j) values. If the FLUSH_LAST_ROW is active,
 * 						the busH will receive the H(m,j) and F(m,j) values, where
 * 						$m$ is the last row of the whole DP matrix. Furthermore,
 * 						when the FLUSH_LAST_ROW is active, the H(m,j) and F(m,j)
 * 						values will be stored in the busH[m-THREAD_COUNT] cell,
 * 						with THREAD_COUNT cells shifted to the left.
 * @param[out] extraH	Extra horizontal bus used if the FLUSH_LAST_ROW is active.
 * 						This is an auxiliary structure that receives
 * 						the values of the horizontal bus that may have an negative
 * 						index (since it is shifted THREAD_COUNT cells to the left).
 * @param[in] h 		The values of H(i,j)..H(i+3,j). This is only used when
 * 						FLUSH_LAST_ROW is active, since we must flush only
 * 						the last row of the DP matrix. Otherwise, only the
 * 						H(i+3,j) will be flushed.
 * @param[in] f			The value of F(i+3,j).
 * @param[in] last_thread	The index of the last thread of the block.
 * @param[in] flush_id		The index of the last DP row in the thread. See function get_flush_id.
 */
template <bool FLUSH_LAST_ROW>
__device__ void kernel_flush(const int i, const int j, const int idx, const int bank, int2* busH, int2* extraH, const int4* h, const int f, const int last_thread, const int flush_id) {
	if (FLUSH_LAST_ROW && flush_id > 0) {
		int h00;
		if (flush_id == 1) {
			h00 = h->x;
		} else if (flush_id == 2) {
			h00 = h->y;
		} else if (flush_id == 3) {
			h00 = h->z;
		} else if (flush_id == 4) {
			h00 = h->w;
		}

		/*
		 * The last row must be shifted THREADS_COUNT bytes to the left in
		 * order to prevent the overwrite of the previous special rows (i.e
		 * since the busH structure is shared by all the blocks, the
		 * bottom-most blocks may overwrite the result of the other blocks).
		 * Since the busH may not be written in negative index,
		 * the extraH vector is used to store this out-of-bound cells.
		 * The remaining cells are written directly to the busH.
		 */
		int2 temp = make_int2(h00, f);
		int adj = j-THREADS_COUNT;
		if (adj < d_split[0]) {
			extraH[j-d_split[0]] = temp;
		} else {
			busH[adj] = temp;
		}
	} else if (idx == last_thread) {
		int2 temp = make_int2(h->w, f);
        busH[j] = temp; // Store into the busH global vector.
    } else {
    	// Store into the shared memory.
        s_colx[bank][idx+1] = h->w;
        s_coly[bank][idx+1] = f;
    }
}


/**
 * After incrementing the column j, this procedure must be called to check if $j$
 * overflows the edge of the sequence#1 (horizontal). If this happens, we must
 * set $j$ to zero and continue the computation in the proper line. In this situation,
 * all registers must be reinitialized in order to represent the first column
 * of the matrix. If the COLUMN_DESTINATION is set with TO_VECTOR, the last
 * column is stored in memory.
 *
 * The in/out params will only be updated if the overflow occurs. Note that
 * every time an overflow happens, the updated cells will consider the kind
 * of alignment (global/semi-global/local) and the source/destination of the values
 * in the edges of the matrix (vector, gap patter or zeroes). See the COLUMN_SOURCE
 * templated variable.
 *
 * @tparam COLUMN_SOURCE 	Indicates how to load the first column. Possible values are:
 * 							FROM_NULL: First column is all zeroed (local alignment)
 * 							FROM_VECTOR: First column is loaded from the loadColumn_h/loadColumn_e vectors.
 * @tparam COLUMN_DESTINATION 	Indicates how to save the last column. Possible values are:
 * 							TO_NULL: Last column is ignored
 * 							TO_VECTOR: Last column is saved into flushColumn_h/flushColumn_e vectors.
 *
 * @param[in] i0	the first row of the DP matrix
 * @param[in] j0	the first column of the DP matrix
 * @param[in] i1	the last row of the DP matrix
 * @param[in] j1	the last column of the DP matrix
 * @param[in,out] i		Input: the current row.
 * 						Output: the updated row if the overflow occurred.
 * @param[in,out] j		Input: the current column.
 * 						Output: the updated column if the overflow occurred.
 * @param[in,out] ss    Input: the variable containing 4 nucleotides of sequence#0.
 * 						Output: the next 4 nucleotides if the overflow occurred.
 * @param[in,out] ee 	Input: the values E(i,j)...E(i+3,j)
 * 						Output: the values E(i,0)...E(i+3,0) if the overflow occurred.
 * @param[in,out] h10 	Input: the values H(i-1,j)
 * 						Output: the values H(i-1,0) if the overflow occurred.
 * @param[in,out] h00 	Input: the values H(i,j)...H(i+3,j)
 * 						Output: the values H(i,0)...H(i+3,0) if the overflow occurred.
 * @param[in,out] flush_id	Input: The index of the last DP row in the thread. See function get_flush_id.
 * 							Output: the updated flush_id if the overflow occurred.
 * @param[in] 	loadColumn_h,loadColumn_e	the values of the first column if COLUMN_SOURCE==FROM_VECTOR.
 * @param[out]  flushColumn_h,flushColumn_e	the destination of the last column if COLUMN_DESTINATION==TO_VECTOR.
 * @param[in]   idx			Index of the current thread.
 * @param[in] 	HEIGHT 		how many lines must be jumped if the overflow occurs.
 */
template <int COLUMN_SOURCE, int COLUMN_DESTINATION>
__device__ void kernel_check_bound(const int i0, const int j0, const int i1, const int j1, int *i, int *j,
        uchar4* ss, int4 *ee, int *h10,  int4 *h00, int *flush_id,
		const int4* loadColumn_h, const int4* loadColumn_e,
		int4* flushColumn_h, int4* flushColumn_e,
		const int idx, const int HEIGHT) {
    if (*j>=d_split[gridDim.x]) {
		if (COLUMN_DESTINATION == STORE_LAST_COLUMN) {
            flushColumn_h[idx] = *h00;
            flushColumn_e[idx] = *ee;
		}

		*j=d_split[0];
        *i+=HEIGHT;

        *flush_id = get_flush_id(*i, i0, i1);

        if (COLUMN_SOURCE == FROM_ZEROES) {
			*ee=make_int4(-INF,-INF,-INF,-INF);
			*h00=make_int4(0,0,0,0);
			*h10=0;
		} else if (COLUMN_SOURCE == FROM_VECTOR) {
			*h10 = loadColumn_h[idx].w;
			*h00 = loadColumn_h[idx+1];
			*ee = loadColumn_e[idx+1];
		}
		fetchSeq0(*i, ss);
    }
}


/**
 * This function processes all the THREADS_COUNT-1 internal diagonals of the short phase.
 *
 * @tparam COLUMN_SOURCE, COLUMN_DESTINATION, RECURRENCE_TYPE, CHECK_LOCATION
 * 				See these templates in the "Detailed Description" section in the beginning of this file.
 *
 * @param[in] idx	Index of the current thread in the block.
 * @param[in] tidx	Index of the current thread in the grid.
 * @param[in] i		The index of the row (top cell).
 * @param[in] j		The index of the column.
 * @param[in] i0,i1	the first and last row of the DP matrix
 * @param[in] j0,j1	the first and last column of the DP matrix
 * @param[in,out] busH		Horizontal bus used to transfer data between blocks (top-down).
 * @param[out] extraH		Extra Horizontal bus. See kernel_flush function for more information.
 * @param[in,out] busV_h,busV_e,busV_o		Vertical bus used to transfer data between blocks (left-right).
 * @param[in,out] max,max_pos	Best score and its position.
 * @param[in] flush_id	The index of the last DP row in the thread. See function get_flush_id.
 * @param[in] loadColumn_h,loadColumn_e		The values of the first column (when COLUMN_SOURCE=FROM_VECTOR).
 * @param[out] flushColumn_h,flushColumn_e		Stores the last column (when COLUMN_DESTINATION=TO_VECTOR).
 */
template<int COLUMN_SOURCE, int COLUMN_DESTINATION, int RECURRENCE_TYPE, int CHECK_LOCATION, bool FLUSH_LAST_ROW>
__device__ void process_internal_diagonals_short_phase(
		const int idx, const int tidx, int i, int j,
		const int i0, const int j0, const int i1, const int j1,
		int2* busH, int2* extraH,
		int4* busV_h, int4* busV_e, int3* busV_o,
		int* max, int2* max_pos,
		int flush_id,
		const int4* loadColumn_h, const int4* loadColumn_e,
		int4* flushColumn_h, int4* flushColumn_e)
{
    s_colx[0][idx] = s_colx[1][idx] = busV_o[tidx].x; // TODO poderia ser pego de busV_h[tidx].w. Certo?
    s_coly[0][idx] = s_coly[1][idx] = busV_o[tidx].y;

    int4 left_h = busV_h[tidx];
	int4 left_e = busV_e[tidx];
    int  diag_h = busV_o[tidx].z;

    uchar4 ss;
	fetchSeq0(i, &ss);

    __syncthreads(); // barrier


	/*
	 *  We need THREADS_COUNT-1 Steps to complete the pending cells.
	 */

	int _k = (THREADS_COUNT>>1)-1; // we divide per 2 because we are loop-unrolling. THREADS_COUNT must be even.
	for (; _k; _k--) {
		int4 curr_h;
		int up_h;
		int up_f;

		/* Loop-unrolling #1 */
		kernel_check_bound<COLUMN_SOURCE, COLUMN_DESTINATION>(i0, d_split[0], i1, d_split[gridDim.x], &i, &j, &ss, &left_e, &diag_h, &left_h, &flush_id, loadColumn_h, loadColumn_e, flushColumn_h, flushColumn_e, idx, blockDim.x*gridDim.x*ALPHA);
		if (flush_id) {
			unsigned char s1;
			kernel_load<true>(idx, 1, j, busH, &up_h, &up_f, &s1);
			kernel_sw4<RECURRENCE_TYPE, FLUSH_LAST_ROW>(i, j, s1, ss, &left_e, &up_f, left_h, diag_h , up_h , &curr_h, flush_id);
			kernel_check_max4<CHECK_LOCATION, FLUSH_LAST_ROW>(i, j, max, max_pos, &curr_h, true, flush_id);
			kernel_flush<FLUSH_LAST_ROW>(i, j, idx, 0, busH, extraH, &curr_h, up_f, THREADS_COUNT-1, flush_id);
		}
		j++;
		__syncthreads(); // barrier

		/* Loop-unrolling #2 */
		kernel_check_bound<COLUMN_SOURCE, COLUMN_DESTINATION>(i0, d_split[0], i1, d_split[gridDim.x], &i, &j, &ss, &left_e, &up_h, &curr_h, &flush_id, loadColumn_h, loadColumn_e, flushColumn_h, flushColumn_e, idx, blockDim.x*gridDim.x*ALPHA);
		if (flush_id) {
			unsigned char s1;
			kernel_load<true>(idx, 0, j, busH, &diag_h, &up_f, &s1);
			kernel_sw4<RECURRENCE_TYPE, FLUSH_LAST_ROW>(i, j, s1, ss, &left_e, &up_f, curr_h, up_h , diag_h, &left_h, flush_id);
			kernel_check_max4<CHECK_LOCATION, FLUSH_LAST_ROW>(i, j, max, max_pos, &left_h, true, flush_id);
			kernel_flush<FLUSH_LAST_ROW>(i, j, idx, 1, busH, extraH, &left_h, up_f, THREADS_COUNT-1, flush_id);
		}
		j++;
		__syncthreads(); // barrier
	}

	{
		/* Last iteration is odd, so we put it outside the unrolled loop */
		int4 curr_h;
		int up_h;
		int up_f;
		kernel_check_bound<COLUMN_SOURCE, COLUMN_DESTINATION>(i0, j0, i1, j1, &i, &j, &ss, &left_e, &diag_h, &left_h, &flush_id, loadColumn_h, loadColumn_e, flushColumn_h, flushColumn_e, idx, blockDim.x*gridDim.x*ALPHA);
		if (flush_id) {
			unsigned char s1;
			kernel_load<true>(idx, 1, j, busH, &up_h, &up_f, &s1);
			kernel_sw4<RECURRENCE_TYPE, FLUSH_LAST_ROW>(i, j, s1, ss, &left_e, &up_f, left_h, diag_h, up_h, &curr_h, flush_id);
			kernel_check_max4<CHECK_LOCATION, FLUSH_LAST_ROW>(i, j, max, max_pos, &curr_h, true, flush_id);
			kernel_flush<FLUSH_LAST_ROW>(i, j, idx, 0, busH, extraH, &curr_h, up_f, THREADS_COUNT-1, flush_id);
		}
		j++;
		__syncthreads(); // barrier

		/* We store the result in the vertical bus to be read by the next block */

		kernel_check_bound<COLUMN_SOURCE, COLUMN_DESTINATION>(i0, j0, i1, j1, &i, &j, &ss, &left_e, &up_h, &curr_h, &flush_id, loadColumn_h, loadColumn_e, flushColumn_h, flushColumn_e, idx, blockDim.x*gridDim.x*ALPHA);
		busV_h[tidx] = curr_h;
		busV_e[tidx] = left_e;
		busV_o[tidx].x = s_colx[0][idx];
		busV_o[tidx].y = s_coly[0][idx];
		busV_o[tidx].z = up_h;
	}
}


/**
 * This kernel processes all the internal diagonal during the short phase. The
 * short phase processes the first THREAD_COUNT-1 diagonals and the long phase
 * processes the remaining diagonals. Each internal diagonal are processed
 * by many threads in parallel, but with a synchronous barrier after each
 * internal diagonal.
 *
 * @tparam COLUMN_SOURCE, COLUMN_DESTINATION, RECURRENCE_TYPE, CHECK_LOCATION
 * 				See these templates in the "Detailed Description" section in the beginning of this file.
 *
 * @param[in] i0,i1		the first and last row of the DP matrix
 * @param[in] step		the id of the external diagonal (0-based)
 * @param[in] cutBlock	(cutBlock.x, cutBlock.y) is the block pruning window.
 * @param[in,out] blockResult 	stores the best score and its position for each block.
 * @param[in,out] busH		Horizontal bus used to transfer data between blocks (top-down).
 * @param[out] extraH		Extra Horizontal bus. See kernel_flush function for more information.
 * @param[in,out] busV_h,busV_e,busV_o		Vertical bus used to transfer data between blocks (left-right).
 * @param[in] loadColumn_h,loadColumn_e		The values of the first column (when COLUMN_SOURCE=FROM_VECTOR).
 * @param[out] flushColumn_h,flushColumn_e		Stores the last column (when COLUMN_DESTINATION=TO_VECTOR).
 */
template <int COLUMN_SOURCE, int COLUMN_DESTINATION, int RECURRENCE_TYPE, bool CHECK_MAX_SCORE>
//__launch_bounds__(THREADS_COUNT,MIN_BLOCKS_PER_SM)
__global__ void kernel_short_phase(const int i0, const int i1,
				const int step, const int2 cutBlock, int4 *blockResult,
				int2* busH, int2* extraH,
				int4* busV_h, int4* busV_e, int3* busV_o,
				const int4* loadColumn_h, const int4* loadColumn_e,
				int4* flushColumn_h, int4* flushColumn_e)
{
    int bx = blockIdx.x;
    int by = step-bx;
    if (by < 0) return;

    int idx = threadIdx.x;

    const volatile int x0 = d_split[bx];
    const int xLen = d_split[bx+1] - x0;

    int i=(by*THREADS_COUNT)+idx;
    int tidx = (i % (blockDim.x*gridDim.x));

    // Block Pruning
    bool pruneBlock;
    if (bx != 0) {
    	pruneBlock = (bx < cutBlock.x || bx > cutBlock.y);
    } else {
    	pruneBlock = (cutBlock.x > 0 && cutBlock.y < blockIdx.x);
    }
	if (pruneBlock) {
		return;
	}

	const int j0 = d_split[0];
	const int j1 = d_split[gridDim.x];

    int j=x0-idx;
    i *= ALPHA;
    i += i0;
    if (j <= j0) {
        j += (j1 - j0);
        i -= (blockDim.x*gridDim.x)*ALPHA;
    }

    int2 max_pos;
    max_pos.x = blockResult[blockIdx.x].x;
    max_pos.y = blockResult[blockIdx.x].y;
    int max = blockResult[blockIdx.x].w;

	//int flush_id = get_flush_id(i, i0, i1);

        if (i >= i1) return;

	if (i < i1) {
		int block_i = (by*THREADS_COUNT)*ALPHA + i0;
		const int flush_id = get_flush_id(i, i0, i1); // TODO why this line is not inside the else clause (see long phase)? Check this.
		if (block_i+THREADS_COUNT*ALPHA < i1) {
			// If the block is fully inside the range [i0..i1], then we do not request to flush the last row.
			process_internal_diagonals_short_phase<COLUMN_SOURCE, COLUMN_DESTINATION, RECURRENCE_TYPE, CHECK_MAX_SCORE, false>(idx, tidx, i, j, i0, j0, i1, j1, busH, extraH, busV_h, busV_e, busV_o, &max, &max_pos, flush_id, loadColumn_h, loadColumn_e, flushColumn_h, flushColumn_e);
		} else {
			// Otherwise, we must flush the last row
			process_internal_diagonals_short_phase<COLUMN_SOURCE, COLUMN_DESTINATION, RECURRENCE_TYPE, CHECK_MAX_SCORE, true> (idx, tidx, i, j, i0, j0, i1, j1, busH, extraH, busV_h, busV_e, busV_o, &max, &max_pos, flush_id, loadColumn_h, loadColumn_e, flushColumn_h, flushColumn_e);
		}
    }


	/* Updates the block result with the block best score */

	if (CHECK_MAX_SCORE) {
		if (findMax(idx, max)) {
			blockResult[blockIdx.x].w = max;
			blockResult[blockIdx.x].x = max_pos.x;
			blockResult[blockIdx.x].y = max_pos.y;
			if (blockResult[blockIdx.x].z < max) {
				blockResult[blockIdx.x].z = max;
			}
		}
	} else {
    	blockResult[blockIdx.x].w = -INF;
	}
}

/**
 * This function processes all the internal diagonals of the long phase.
 *
 * @tparam RECURRENCE_TYPE, CHECK_LOCATION, FLUSH_LAST_ROW
 * 				See these templates in the "Detailed Description" section in the beginning of this file.
 *
 * @param[in] xLen	Number of internal diagonals.
 * @param[in] idx	Index of the current thread in the block.
 * @param[in] tidx	Index of the current thread in the grid.
 * @param[in] i		The index of the row (top cell).
 * @param[in] j		The index of the column.
 * @param[in,out] busH		Horizontal bus used to transfer data between blocks (top-down).
 * @param[out] extraH		Extra Horizontal bus. See kernel_flush function for more information.
 * @param[in,out] busV_h,busV_e,busV_o		Vertical bus used to transfer data between blocks (left-right).
 * @param[in,out] max,max_pos	Best score and its position.
 * @param[in] flush_id	The index of the last DP row in the thread. See function get_flush_id.
 */
template<int RECURRENCE_TYPE, int CHECK_LOCATION, bool FLUSH_LAST_ROW>
__device__ void process_internal_diagonals_long_phase(const int xLen, const int idx, const int tidx,
		const int i, int j,
		int2* busH, int2* extraH,
		int4* busV_h, int4* busV_e, int3* busV_o,
		int* max, int2* max_pos,
		const int flush_id)
{

    s_colx[0][idx] = s_colx[1][idx] = busV_o[tidx].x;
    s_coly[0][idx] = s_coly[1][idx] = busV_o[tidx].y;

    int4 left_h = busV_h[tidx];
	int4 left_e = busV_e[tidx];
    int  diag_h = busV_o[tidx].z;

    uchar4 ss;
	fetchSeq0(i, &ss);

    __syncthreads();


	int _k = xLen;
	if (_k&1) { // if odd
		int4 cur_h;
		int up_h;
		int up_f;

		unsigned char s1;
		kernel_load<true>(idx, 1, j, busH, &up_h, &up_f, &s1);
		kernel_sw4<RECURRENCE_TYPE, FLUSH_LAST_ROW>(i, j, s1, ss, &left_e, &up_f, left_h, diag_h, up_h, &cur_h, flush_id);
		kernel_check_max4<CHECK_LOCATION, FLUSH_LAST_ROW>(i, j, max, max_pos, &cur_h, false, flush_id);
		kernel_flush<FLUSH_LAST_ROW>(i, j, idx, 0, busH, extraH, &cur_h, up_f, THREADS_COUNT-1, flush_id);
		j++;
		__syncthreads();
		s_colx[1][idx] = s_colx[0][idx];
		s_coly[1][idx] = s_coly[0][idx];
		diag_h = up_h;
		left_h = cur_h;
		__syncthreads();
		_k--;
	}
	_k >>= 1; // we divide per 2 because we are loop-unrolling.
	for (; _k; _k--) {
		int4 cur_h;
		int up_h;
		int up_f;

		unsigned char s1;

		/* Loop-unrolling #1 */
		kernel_load<true>(idx, 1, j, busH, &up_h, &up_f, &s1);
		kernel_sw4<RECURRENCE_TYPE, FLUSH_LAST_ROW>(i, j, s1, ss, &left_e, &up_f, left_h, diag_h, up_h, &cur_h, flush_id);
		kernel_check_max4<CHECK_LOCATION, FLUSH_LAST_ROW>(i, j, max, max_pos, &cur_h, false, flush_id);
		kernel_flush<FLUSH_LAST_ROW>(i, j, idx, 0, busH, extraH, &cur_h, up_f, THREADS_COUNT-1, flush_id);
		j++;
		__syncthreads();

		/* Loop-unrolling #2 */
		kernel_load<true>(idx, 0, j, busH, &diag_h, &up_f, &s1);
		kernel_sw4<RECURRENCE_TYPE, FLUSH_LAST_ROW>(i, j, s1, ss, &left_e, &up_f, cur_h, up_h, diag_h, &left_h, flush_id);
		kernel_check_max4<CHECK_LOCATION, FLUSH_LAST_ROW>(i, j, max, max_pos, &left_h, false, flush_id);
		kernel_flush<FLUSH_LAST_ROW>(i, j, idx, 1, busH, extraH, &left_h, up_f, THREADS_COUNT-1, flush_id);
		j++;
		__syncthreads();

	}

	/* We store the result in the vertical bus to be read by the next block */

    busV_h[tidx]=left_h;
    busV_e[tidx]=left_e;
    busV_o[tidx].x=s_colx[1][idx];
    busV_o[tidx].y=s_coly[1][idx];
    busV_o[tidx].z=diag_h;

}

/**
 * This kernel processes all the internal diagonal during the long phase. The
 * short phase processes the first THREAD_COUNT-1 diagonals and the long phase
 * processes the remaining diagonals. Each internal diagonal are processed
 * by many threads in parallel, but with a synchronous barrier after each
 * internal diagonal.
 *
 * @tparam RECURRENCE_TYPE, CHECK_LOCATION
 * 				See these templates in the "Detailed Description" section in the beginning of this file.
 *
 * @param[in] i0,i1		the first and last row of the DP matrix
 * @param[in] step		the id of the external diagonal (0-based)
 * @param[in] cutBlock	(cutBlock.x, cutBlock.y) is the pruning window.
 * @param[in,out] blockResult 	stores the best score and its position for each block.
 * @param[in,out] busH		Horizontal bus used to transfer data between blocks (top-down).
 * @param[out] extraH		Extra Horizontal bus. See kernel_flush function for more information.
 * @param[in,out] busV_h,busV_e,busV_o		Vertical bus used to transfer data between blocks (left-right).
 */
//__launch_bounds__(THREADS_COUNT,MIN_BLOCKS_PER_SM)
// TODO testar com template FLUSH_LAST_ROW, pois ficou mais lento!
template <int RECURRENCE_TYPE, bool CHECK_MAX_SCORE>
__global__ void kernel_long_phase(
		const int i0, const int i1,
		const int step,
		const int2 cutBlock, int4 *blockResult,
		int2* busH, int2* extraH,
		int4* busV_h, int4* busV_e, int3* busV_o)
{
	const int bx = blockIdx.x;
	if (bx < cutBlock.x || bx > cutBlock.y) {
		// Block Pruning
		if (step - bx >= 0) {
			int tidx = ((step-bx) % gridDim.x)*THREADS_COUNT + threadIdx.x;
			busV_h[tidx]=make_int4(-INF,-INF,-INF,-INF);
			busV_e[tidx]=make_int4(-INF,-INF,-INF,-INF);
			busV_o[tidx]=make_int3(-INF,-INF,-INF);
			blockResult[bx].w = -INF;
		}
		return;
	}
	const int by = step-bx;
    if (by < 0) return;

    const int idx = threadIdx.x;

    const int x0 = d_split[bx]+(THREADS_COUNT-1);
    const int xLen = d_split[bx+1] - x0;

    const int tidx = (by % gridDim.x)*THREADS_COUNT+idx;
    const int i=((by*THREADS_COUNT)+idx)*ALPHA + i0;

    //const int j1 = d_split[gridDim.x];

    const int j=x0-idx;

    int2 max_pos;
    max_pos.x = -1;//blockResult[blockIdx.x].x;
    max_pos.y = -1;//blockResult[blockIdx.x].y;
    int max = -INF;//blockResult[bx].w;

        if (i >= i1) return;
       
	if (i < i1) {
		const int block_i = (by*THREADS_COUNT)*ALPHA + i0;
		if (block_i+THREADS_COUNT*ALPHA < i1) {
			// If the block is fully inside the range [i0..i1], then we do not request to flush the last row.
			process_internal_diagonals_long_phase<RECURRENCE_TYPE, CHECK_MAX_SCORE, false>(xLen, idx, tidx, i, j, busH, extraH, busV_h, busV_e, busV_o, &max, &max_pos, -1);
		} else {
			// Otherwise, we must flush the last row
			const int flush_id = get_flush_id(i, i0, i1);
			process_internal_diagonals_long_phase<RECURRENCE_TYPE, CHECK_MAX_SCORE, true >(xLen, idx, tidx, i, j, busH, extraH, busV_h, busV_e, busV_o, &max, &max_pos, flush_id);
		}
    }

	if (CHECK_MAX_SCORE) {
		if (findMax(threadIdx.x, max)) {
			blockResult[blockIdx.x].w = max;
			blockResult[blockIdx.x].x = max_pos.x;
			blockResult[blockIdx.x].y = max_pos.y+i;
			if (blockResult[blockIdx.x].z < max) {//max_pos.x != -1) {
				blockResult[blockIdx.x].z = max;
			}
		}
	} else {
    	blockResult[blockIdx.x].w = -INF;
	}
}

/**
 * This function processes the internal diagonals of the single phase (very small sequences).
 *
 * @tparam COLUMN_SOURCE, COLUMN_DESTINATION, RECURRENCE_TYPE, CHECK_LOCATION, FLUSH_LAST_ROW
 * 				See these templates in the "Detailed Description" section in the beginning of this file.
 *
 * @param[in] xLen	Number of internal diagonals.
 * @param[in] idx	Index of the current thread in the single block.
 * @param[in] i		The index of the row (top cell).
 * @param[in] j		The index of the column.
 * @param[in] i0,i1	the first and last row of the DP matrix
 * @param[in] j0,j1	the first and last column of the DP matrix
 * @param[in,out] busH		Horizontal bus used to transfer data between blocks (top-down).
 * @param[out] extraH		Extra Horizontal bus. See kernel_flush function for more information.
 * @param[in,out] busV_h,busV_e,busV_o		Vertical bus used to transfer data between blocks (left-right).
 * @param[in,out] max,max_pos	Best score and its position.
 * @param[in] flush_id	The index of the last DP row in the thread. See function get_flush_id.
 * @param[in] loadColumn_h,loadColumn_e		The values of the first column (when COLUMN_SOURCE=FROM_VECTOR).
 * @param[out] flushColumn_h,flushColumn_e		Stores the last column (when COLUMN_DESTINATION=TO_VECTOR).
 */
template<int COLUMN_SOURCE, int COLUMN_DESTINATION, int RECURRENCE_TYPE, int CHECK_LOCATION, bool FLUSH_LAST_ROW>
__device__ void process_internal_diagonals_single_phase(
		const int xLen, const int idx,
		int i, int j, const int i0, const int j0, const int i1, const int j1,
		int2* busH, int2* extraH,
		int4* busV_h, int4* busV_e, int3* busV_o,
		int* max, int2* max_pos,
		int flush_id,
		const int4* loadColumn_h, const int4* loadColumn_e,
		int4* flushColumn_h, int4* flushColumn_e)
{
    s_colx[0][idx] = s_colx[1][idx] = busV_o[idx].x;
    s_coly[0][idx] = s_coly[1][idx] = busV_o[idx].y;

	int4 left_h = busV_h[idx];
	int4 left_e = busV_e[idx];
    int  diag_h = busV_o[idx].z;

    uchar4 ss;
	fetchSeq0(i, &ss);

    __syncthreads();

    int _k = xLen;
    int index = 1;
    for (; _k; _k--) {
    	int4 curr_h;
        int  up_h;
        int  up_f;
		kernel_check_bound<COLUMN_SOURCE, COLUMN_DESTINATION>(i0, j0, i1, j1, &i, &j, &ss, &left_e, &diag_h, &left_h, &flush_id, loadColumn_h, loadColumn_e, flushColumn_h, flushColumn_e, idx, blockDim.x*ALPHA);
        if (flush_id) {
            unsigned char s1;
            kernel_load<false>(idx, index, j, busH, &up_h, &up_f, &s1);
			kernel_sw4<RECURRENCE_TYPE, FLUSH_LAST_ROW>(i, j, s1, ss, &left_e, &up_f, left_h, diag_h  , up_h  , &curr_h, flush_id);
			kernel_check_max4<CHECK_LOCATION, FLUSH_LAST_ROW>(i, j, max, max_pos, &curr_h, true, flush_id);
            kernel_flush<FLUSH_LAST_ROW>(i, j, idx, 1-index, busH, extraH, &curr_h, up_f, blockDim.x-1, flush_id);
        }
        index = 1-index;
        j++;
        diag_h = up_h;
        left_h = curr_h;
        __syncthreads();
    }
    kernel_check_bound<COLUMN_SOURCE, COLUMN_DESTINATION>(i0, j0, i1, j1, &i, &j, &ss, &left_e, &diag_h, &left_h, &flush_id, loadColumn_h, loadColumn_e, flushColumn_h, flushColumn_e, idx, blockDim.x*ALPHA);

    busV_h[idx]=left_h;
    busV_e[idx]=left_e;
    busV_o[idx].x=s_colx[index][idx];
    busV_o[idx].y=s_coly[index][idx];
    busV_o[idx].z=diag_h;

}

/**
 * This kernel processes all the internal diagonal with a single phase.
 * The single phase is used for very small sequences and only one block must
 * be executed.
 *
 * @tparam COLUMN_SOURCE, COLUMN_DESTINATION, RECURRENCE_TYPE, CHECK_LOCATION
 * 				See these templates in the "Detailed Description" section in the beginning of this file.
 *
 * @param[in] i0,i1		the first and last row of the DP matrix
 * @param[in] step		the id of the external diagonal (0-based)
 * @param[in] cutBlock	(cutBlock.x, cutBlock.y) is the pruning window.
 * @param[in,out] blockResult 	stores the best score and its position for each block.
 * @param[in,out] busH		Horizontal bus used to transfer data between external diagonals.
 * @param[out] extraH		Extra Horizontal bus. See kernel_flush function for more information.
 * @param[in,out] busV_h,busV_e,busV_o		Vertical bus used to transfer data between external diagonals.
 * @param[in] loadColumn_h,loadColumn_e		The values of the first column (when COLUMN_SOURCE=FROM_VECTOR).
 * @param[out] flushColumn_h,flushColumn_e		Stores the last column (when COLUMN_DESTINATION=TO_VECTOR).
 */
template <int COLUMN_SOURCE, int COLUMN_DESTINATION, int RECURRENCE_TYPE, int CHECK_LOCATION>
//__launch_bounds__(THREADS_COUNT,1)
__global__ void kernel_single_phase(
		const int i0, const int i1,
		const int step, const int2 cutBlock, int4 *blockResult,
		int2* busH, int2* extraH,
		int4* busV_h, int4* busV_e, int3* busV_o,
		const int4* loadColumn_h, const int4* loadColumn_e,
		int4* flushColumn_h, int4* flushColumn_e)
{

    int idx = threadIdx.x;
    int by = step;

	const int j0 = d_split[0];
	const int j1 = d_split[gridDim.x];

    const int xLen = j1-j0;
    int j=j0-idx-(xLen-blockDim.x+1); // This ensures that the block fills the busH entirely in the same row
    int i=(by*blockDim.x)+idx;
    i *= ALPHA;
    i += i0;

    if (j<=j0) { // TODO era while em vez de if
        j+=(j1-j0);
        i-=blockDim.x*ALPHA;
    }

    int2 max_pos;
    max_pos.x = -1;//blockResult[0].x;
    max_pos.y = -1;//blockResult[0].y;
    int max = -INF;//blockResult[0].w;

    int flush_id = get_flush_id(i, i0, i1);
	// TODO otimizar considerando warps
	if (i >= i1) return;

        if (i < i1) {
		const int block_i = (by*blockDim.x)*ALPHA +i0;
		//const int flush_id = get_flush_id(i, i0, i1);
		if (block_i+blockDim.x*ALPHA < i1) {
			// If the block is fully inside the range [i0..i1], then we do not request to flush the last row.
			process_internal_diagonals_single_phase<COLUMN_SOURCE, COLUMN_DESTINATION, RECURRENCE_TYPE, CHECK_LOCATION, false>(xLen, idx, i, j, i0, j0, i1, j1, busH, extraH, busV_h, busV_e, busV_o, &max, &max_pos, flush_id, loadColumn_h, loadColumn_e, flushColumn_h, flushColumn_e);
		} else {
			process_internal_diagonals_single_phase<COLUMN_SOURCE, COLUMN_DESTINATION, RECURRENCE_TYPE, CHECK_LOCATION, true>(xLen, idx, i, j, i0, j0, i1, j1, busH, extraH, busV_h, busV_e, busV_o, &max, &max_pos, flush_id, loadColumn_h, loadColumn_e, flushColumn_h, flushColumn_e);
		}
	}

    if (findMaxSmall(idx, max)) {
		blockResult[0].w = max;
		blockResult[0].x = max_pos.x;
		blockResult[0].y = max_pos.y;
        if (blockResult[0].z < max) {//max_pos.x != -1) {
            blockResult[0].z = max;
        }
    }
}

/**
 * Initialize the horizontal bus considering a zeroed first row (local alignment)
 * or an -infinity first row (prune block).
 *
 * @tparam H,F	the value of the H and F components to set in each row.
 *
 * @param[out] busH		horizontal bus.
 * @param[in] j0		first column to be processed.
 * @param[in] len		length of the range to be initialized (j0, j0+len).
 */
template<int H, int F>
static __global__ void kernel_initialize_busH_ungapped(int2* busH, const int j0, const int len) {
    int tidx = blockIdx.x*blockDim.x + threadIdx.x;
    while (tidx < len) {
		busH[j0+tidx].x = H;
        busH[j0+tidx].y = F;

		tidx += blockDim.x*gridDim.x;
    }
}

/**
 * Bind the textures for the DNA sequences.
 * @param seq0 sequence 0
 * @param seq0_len sequence 0 length
 * @param seq1 sequence 1
 * @param seq1_len sequence 1 length
 */
void bind_textures(const unsigned char* seq0, const int seq0_len, const unsigned char* seq1, const int seq1_len) {
	cutilSafeCall(hipBindTexture(0, t_seq0, seq0, seq0_len));
	cutilSafeCall(hipBindTexture(0, t_seq1, seq1, seq1_len));
}

/**
 * Unbind the textures for the DNA sequences.
 */
void unbind_textures() {
	cutilSafeCall(hipUnbindTexture(t_seq1));
	cutilSafeCall(hipUnbindTexture(t_seq0));
}

/**
 * Copies the split positions (used to identify the range of columns for each
 * block) to the GPU constant memory. The element split[0] must be the first
 * column of the partition and split[blocks] must be the last column.
 *
 * @param split vector with the block split positions.
 * @param blocks number of blocks.
 */
void copy_split(const int* split, const int blocks) {
	cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_split), split, (blocks+1)*sizeof(int)));
}

/**
 * Initialize the horizontal bus with H=-INF and F=-INF.
 *
 * @param p0,p1 range to be initialized.
 * @param d_busH bus to be initialized.
 */
void initializeBusHInfinity(const int p0, const int p1, int2* d_busH) {
	dim3 threads(512, 1, 1);
	dim3 blocks(MAX_BLOCKS_COUNT, 1, 1);
	kernel_initialize_busH_ungapped<-INF,-INF><<<threads, blocks>>>(d_busH, p0, p1-p0);
	cutilCheckMsg("Kernel execution failed");
}
//void CUDAligner::initializeBusHUngapped(const int p0, const int p1) {
//	dim3 threads(512, 1, 1);
//	dim3 blocks(BLOCKS_COUNT, 1, 1);
//	kernel_initialize_busH_ungapped<0,-INF><<<threads, blocks>>>(cuda.d_busH, p0, p1-p0);
//	cutilCheckMsg("Kernel execution failed");
//}

/****************************************************************************
 *
 * Templates function for creating precompiled different kernels.
 * Since we have four templates (COLUMN_SOURCE, COLUMN_DESTINATION,
 * RECURRENCE_TYPE, CHECK_LOCATION) with many options, we have
 * a total of 2x2x2x2 = 16 precompiled kernels with different
 * constant parameters. This improve significantly the
 * performance of the different kernel calls, but increases
 * compilation time and executable size.
 *
 ****************************************************************************/

/**
 * Invoke a kernel to process one external diagonal. Depending on the number
 * of blocks, it decides which kernel will be called: single phase or
 * short/long phase. The templates creates many precompiled kernels with
 * different constant parameters, instead of variable parameters. This increases
 * the performance with a tradeoff of creating 16 precompiled kernels (bigger
 * executable size).
 *
 * @tparam COLUMN_SOURCE, COLUMN_DESTINATION, RECURRENCE_TYPE, CHECK_LOCATION
 * 				See these templates in the "Detailed Description" section in the beginning of this file.
 * @param blocks Number of blocks. If blocks is equal to 1, then the
 * single phase kernel is called, otherwise it calls the short/long kernels.
 * @param threads Maximum number of threads per block.
 * @param i0 first column id.
 * @param i1 last column id.
 * @param step the current external diagonal id, starting from 0.
 * @param cutBlock pruning window.
 * @param cuda the object containing all the cuda allocated structures.
 */
template <int COLUMN_SOURCE, int COLUMN_DESTINATION, int RECURRENCE_TYPE, int CHECK_LOCATION>
void lauch_external_diagonals(const int blocks, const int threads,
		const int i0, const int i1,
		const int step, const int2 cutBlock, cuda_structures_t* cuda) {
	cutilSafeCall(hipBindTexture(0, t_busH, cuda->d_busH, cuda->busH_size));
	dim3 grid( blocks, 1, 1);
	if (blocks == 1) {
		dim3 block( threads, 1, 1);
		kernel_single_phase<COLUMN_SOURCE, COLUMN_DESTINATION, RECURRENCE_TYPE, CHECK_LOCATION><<< grid, block, 0>>>(i0, i1, step, cutBlock, cuda->d_blockResult, cuda->d_busH, cuda->d_extraH, cuda->d_busV_h, cuda->d_busV_e, cuda->d_busV_o, cuda->d_loadColumnH, cuda->d_loadColumnE, cuda->d_flushColumnH, cuda->d_flushColumnE);
	} else {
		static dim3 block( THREADS_COUNT, 1, 1);
		kernel_long_phase<RECURRENCE_TYPE, CHECK_LOCATION><<< grid, threads, 0>>>(i0, i1, step-1, cutBlock, cuda->d_blockResult, cuda->d_busH, cuda->d_extraH, cuda->d_busV_h, cuda->d_busV_e, cuda->d_busV_o);
		kernel_short_phase<COLUMN_SOURCE, COLUMN_DESTINATION, RECURRENCE_TYPE, CHECK_LOCATION><<< grid, threads, 0>>>(i0, i1, step, cutBlock, cuda->d_blockResult, cuda->d_busH, cuda->d_extraH, cuda->d_busV_h, cuda->d_busV_e, cuda->d_busV_o, cuda->d_loadColumnH, cuda->d_loadColumnE, cuda->d_flushColumnH, cuda->d_flushColumnE);
	}
	hipStreamSynchronize(0);
	cutilCheckMsg("Kernel execution failed");
	cutilSafeCall(hipUnbindTexture(t_busH));
}


/* Templated-inline Function */
template <int COLUMN_SOURCE, int COLUMN_DESTINATION, int RECURRENCE_TYPE>
void lauch_external_diagonals(int CHECK_LOCATION, const int blocks, const int threads,
		const int i0, const int i1,
		const int step, const int2 cutBlock, cuda_structures_t* cuda) {
	if (CHECK_LOCATION) {
		lauch_external_diagonals<COLUMN_SOURCE, COLUMN_DESTINATION, RECURRENCE_TYPE, CHECK_BEST_SCORE>(blocks, threads, i0, i1, step, cutBlock, cuda);
	} else {
		lauch_external_diagonals<COLUMN_SOURCE, COLUMN_DESTINATION, RECURRENCE_TYPE, IGNORE_BEST_SCORE>(blocks, threads, i0, i1, step, cutBlock, cuda);
	}
}

/* Templated-inline Function */
template <int COLUMN_SOURCE, int COLUMN_DESTINATION>
void lauch_external_diagonals(int RECURRENCE_TYPE, int CHECK_LOCATION, const int blocks, const int threads,
		const int i0, const int i1,
		const int step, const int2 cutBlock, cuda_structures_t* cuda) {
	if (RECURRENCE_TYPE == SMITH_WATERMAN) {
		lauch_external_diagonals<COLUMN_SOURCE, COLUMN_DESTINATION, SMITH_WATERMAN>(CHECK_LOCATION, blocks, threads, i0, i1, step, cutBlock, cuda);
	} else if (RECURRENCE_TYPE == NEEDLEMAN_WUNSCH) {
		lauch_external_diagonals<COLUMN_SOURCE, COLUMN_DESTINATION, NEEDLEMAN_WUNSCH>(CHECK_LOCATION, blocks, threads, i0, i1, step, cutBlock, cuda);
	} else {
		// DIE
	}
}


/* Templated-inline Function */
template <int COLUMN_SOURCE>
void lauch_external_diagonals(int COLUMN_DESTINATION, int RECURRENCE_TYPE, int CHECK_LOCATION, const int blocks, const int threads,
		const int i0, const int i1,
		const int step, const int2 cutBlock, cuda_structures_t* cuda) {
	if (COLUMN_DESTINATION) {
		lauch_external_diagonals<COLUMN_SOURCE, STORE_LAST_COLUMN>(RECURRENCE_TYPE, CHECK_LOCATION, blocks, threads, i0, i1, step, cutBlock, cuda);
	} else {
		lauch_external_diagonals<COLUMN_SOURCE, DISCARD_LAST_COLUMN>(RECURRENCE_TYPE, CHECK_LOCATION, blocks, threads, i0, i1, step, cutBlock, cuda);
	}
}

/* Templated-inline Function */
void lauch_external_diagonals(int COLUMN_SOURCE, int COLUMN_DESTINATION, int RECURRENCE_TYPE, int CHECK_LOCATION, const int blocks, const int threads,
		const int i0, const int i1,
		const int step, const int2 cutBlock, cuda_structures_t* cuda) {
	switch (COLUMN_SOURCE) {
		case INIT_WITH_ZEROES:
			lauch_external_diagonals<FROM_ZEROES>(COLUMN_DESTINATION, RECURRENCE_TYPE, CHECK_LOCATION, blocks, threads, i0, i1, step, cutBlock, cuda);
			break;
		default:
			lauch_external_diagonals<FROM_VECTOR>(COLUMN_DESTINATION, RECURRENCE_TYPE, CHECK_LOCATION, blocks, threads, i0, i1, step, cutBlock, cuda);
			break;
	}
}




